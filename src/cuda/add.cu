
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <stdio.h>

__global__
void add(int *a, int *b, int *c)
{
  c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

#define N 512
int main(void)
{
  int *a, *b, *c;
  int *d_a, *d_b, *d_c;
  int size = N * sizeof(int);

  // Allocate space on the device
  hipMalloc(&d_a, size);
  hipMalloc(&d_b, size);
  hipMalloc(&d_c, size);

  // Inputs
  a = (int *)malloc(size);
  b = (int *)malloc(size);
  c = (int *)malloc(size);

  for (int i = 0; i < N; i++)
  {
    *(a + i) = 2;
    *(b + i) = 4;
  }

  // Copy inputs to device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  // Perform computation on device
  add<<<N,1>>>(d_a, d_b, d_c);

  // Copy result back to host
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
  for (int i = 0; i < N; i++)
  {
    printf("result: %d\n", *c);
  }

  // Clean up
  free(a); free(b); free(c);
  hipFree(d_a); hipFree(d_b); hipFree(d_c);
  return 0;
}
